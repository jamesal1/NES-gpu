#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_fp16.h>


//can't use __clz
int next_pow2(int v) {
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    v++;
    return v;
}

int next_pow2to1024(int v) {
    if (v>512)
        return 1024;
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    v++;
    return v;
}

int ceil_div (int a, int b) {
    return (a + b - 1) / b;

}


__global__ void cuda_pack8_kernel(torch::PackedTensorAccessor32<int8_t,2,torch::RestrictPtrTraits> ret,
                                 torch::PackedTensorAccessor32<bool,2,torch::RestrictPtrTraits> input) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int y_input = y * 8;
    if (x < input.size(0)) {

        int end = input.size(1) - y_input;
        int8_t tmp = 0;
        if (end>7) {
            end = 8;
        }
        int c = 1;
        for (int i = 0; i < end; i++) {
            tmp += c * input[x][y_input+i];
            c*=2;
        }
        ret[x][y] = tmp;
    }
}


//slower than non-templated version for some reason
template <typename scalar_t>
__global__ void cuda_pack_kernel(torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> ret,
                                 torch::PackedTensorAccessor32<bool,2,torch::RestrictPtrTraits> input,
                                 int elementSize) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int y_input = y * elementSize;
    if (x < ret.size(0) && y < ret.size(1)) {
        scalar_t c = 1;
        int end = input.size(1) - y_input;
        if (end > elementSize) {
            end = elementSize;
        }
        scalar_t tmp = 0;
        for (int i = 0; i < end; i++) {
            tmp |= c * input[x][y_input+i];
            c *= 2;
        }
        ret[x][y] = tmp;
    }
}


template <typename scalar_t>
__global__ void cuda_unpack_kernel(torch::PackedTensorAccessor32<bool,2,torch::RestrictPtrTraits> ret,
                               torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> input, int elementSize) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y;
    const int y_output = y * elementSize;
    if (x < input.size(0)) {
            unsigned long long int c = 1;
            for (int i = 0; i < elementSize; i++) {
                ret[x][y_output+i] = (c & input[x][y]) > 0;
                c *= 2;
            }
        }
}


template <typename scalar_t>
__global__ void cuda_binary_bmm_kernel(torch::PackedTensorAccessor32<int32_t,3,torch::RestrictPtrTraits> C,
                               torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> A,
                               torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> B) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (x < C.size(0) && y < C.size(1) && z < C.size(2)) {
        int tmp = 0;
        for (int i = 0; i < A.size(2); i++) {
            tmp += __popcll(A[x][y][i] ^ B[x][i][z]);
        }
        C[x][y][z] = tmp;
    }
}

template <typename scalar_t>
__global__ void cuda_binary_seeded_bmv_kernel(torch::PackedTensorAccessor32<int32_t,2,torch::RestrictPtrTraits> C,
                               torch::PackedTensorAccessor32<torch::Half,2,torch::RestrictPtrTraits> A,
                               torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> B,
                               int elementSize,
                               unsigned long seed) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int ylen = blockDim.y * gridDim.y;
    const int seq = x * ylen + y;
    hiprandState state;
    hiprand_init(seed + seq, 0, 0, &state);
    if (x < C.size(0) && y < C.size(1)) {
        int tmp = 0;
        for (int i = 0; i < B.size(1); i++) {
            scalar_t c = 1;
            int i_bits = i * elementSize;
            int end = A.size(1) - i_bits;
            if (end > elementSize) {
                end = elementSize;
            }
            scalar_t Axyi = 0;
            for (int j = 0; j < end; j++) {
                Axyi |= c * ( __half2float(A[y][i_bits + j]) > hiprand_uniform(&state));
                c *= 2;
            }
            tmp += __popcll(Axyi ^ B[x][i]);
        }
        C[x][y] = tmp;
    }
}


template <typename scalar_t>
__global__ void cuda_sample_bits_kernel(torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> ret,
                                        const torch::PackedTensorAccessor32<torch::Half,2,torch::RestrictPtrTraits> input,
                                        const int elementSize,
                                        const unsigned long seed) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y; //thank whoever made 64 max threads for z
    const int y = blockIdx.z * blockDim.z + threadIdx.z;
    const int zlen = blockDim.y * gridDim.y;
    const int ylen = blockDim.z * gridDim.z;
    const int seq = x * ylen * zlen + y * zlen  + z;
    hiprandState state;
    hiprand_init(seed + seq, 0, 0, &state);
    const int z_input = z * elementSize;
    if (x < ret.size(0) && y < ret.size(1) && z < ret.size(2)) {
        scalar_t c = 1;
        int end = input.size(1) - z_input;
        if (end > elementSize) {
            end = elementSize;
        }
        scalar_t tmp = 0;
        for (int i = 0; i < end; i++) {
            tmp |= c * ( __half2float(input[y][z_input+i]) > hiprand_uniform(&state));
            //tmp |= c * ( (input[y][z_input+i]) > hiprand_normal(&state));
            //tmp |= c * ( (input[y][z_input+i]) > .5);
            c *= 2;
        }
        ret[x][y][z] = tmp;
    }
}

template <typename scalar_t>
__global__ void cuda_binary_weighted_sum_kernel(torch::PackedTensorAccessor32<torch::Half,2,torch::RestrictPtrTraits> ret,
                                    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> input,
                                    torch::PackedTensorAccessor32<torch::Half,1,torch::RestrictPtrTraits> weights,
                                    int elementSize) {
    //const int y = blockIdx.y * blockDim.y + threadIdx.y;
    //const int z = blockIdx.z * blockDim.z + threadIdx.z;
    const int y = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (y < ret.size(0) && z < ret.size(1)) {
        const int z_input = z / elementSize;
        const int z_bit = z - z_input * elementSize;
        scalar_t c = 1;
        c <<= z_bit;
        torch::Half tmp(0);
        for (int i = 0; i < input.size(0); i++) {
            tmp += weights[i] * ((input[i][y][z_input] & c) >> z_bit);
        }
        ret[y][z] = tmp;
    }
}


torch::Tensor cuda_pack8(torch::Tensor input) {
    const int ret_size1 = (input.size(1) + 7) / 8;
    const int threadsy = ret_size1 > 512 ? 1024 : next_pow2(ret_size1);
    const dim3 threads(1024 / threadsy, threadsy);
    const dim3 blocks((input.size(0) + threads.x - 1)/threads.x, ret_size1 / threadsy);
    auto ret = torch::zeros({input.size(0), ret_size1}, torch::TensorOptions().dtype(torch::kInt8).device(input.device()));
    cuda_pack8_kernel<<<blocks,threads>>>(
        ret.packed_accessor32<int8_t,2,torch::RestrictPtrTraits>(),
        input.packed_accessor32<bool,2,torch::RestrictPtrTraits>());
    return ret;
}




torch::Tensor cuda_pack(torch::Tensor input, torch::Dtype dtype) {
    const int bitsize = 8 * elementSize(dtype);
    const int ret_size1 = (input.size(1) + bitsize - 1) / bitsize;
    const int threadsy = ret_size1 > 512 ? 1024 : next_pow2(ret_size1);
    const dim3 threads(1024 / threadsy, threadsy);
    const dim3 blocks((input.size(0) + threads.x - 1)/threads.x, ret_size1 / threadsy);
    auto ret = torch::zeros({input.size(0), ret_size1}, torch::TensorOptions().dtype(dtype).device(input.device()));
    AT_DISPATCH_INTEGRAL_TYPES(ret.scalar_type(), "pack_cuda", ([&] {
        cuda_pack_kernel<<<blocks,threads>>>(
            ret.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
            input.packed_accessor32<bool,2,torch::RestrictPtrTraits>(),
            bitsize);
    }));
    return ret;
}




torch::Tensor cuda_unpack(torch::Tensor input) {
    int bitsize = 8 * elementSize(input.scalar_type());
    const int threads = 1024;
    const dim3 blocks((input.size(0) + threads - 1) / threads, input.size(1));
    auto ret = torch::zeros({input.size(0), input.size(1) * bitsize}, torch::TensorOptions().dtype(torch::kBool).device(input.device()));
    AT_DISPATCH_INTEGRAL_TYPES(input.scalar_type(), "unpack_cuda", ([&] {
        cuda_unpack_kernel<<<blocks,threads>>>(
                    ret.packed_accessor32<bool,2,torch::RestrictPtrTraits>(),
                    input.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
                    bitsize);

    }));
    return ret;
}

torch::Tensor cuda_binary_bmm(torch::Tensor A, torch::Tensor B) {
    auto C = torch::zeros({A.size(0), A.size(1), B.size(2)}, torch::TensorOptions().dtype(torch::kInt32).device(A.device()));
     dim3 threads;
    if (C.size(1) == 1) { //improve later
        threads = dim3(1, 1, 1024);
    } else if (C.size(2) == 1) {
        threads = dim3(1, 1024, 1);
    } else {
        threads = dim3(1, 32, 32);
    }
    //threads=dim3(1024);
    const dim3 blocks((C.size(0) + threads.x - 1) / threads.x, (C.size(1) + threads.y - 1) / threads.y, (C.size(2) + threads.z - 1) / threads.z);
    AT_DISPATCH_INTEGRAL_TYPES(A.scalar_type(), "binary_bmm_cuda", ([&] {
            cuda_binary_bmm_kernel<<<blocks,threads>>>(
                        C.packed_accessor32<int32_t,3,torch::RestrictPtrTraits>(),
                        A.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                        B.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>()
                        );

        }));
    return C;
}

torch::Tensor cuda_binary_seeded_bmv(torch::Tensor A, torch::Tensor B, unsigned long seed) {
    const int bitsize = 8 * elementSize(B.scalar_type());
    auto C = torch::zeros({B.size(0), A.size(0)}, torch::TensorOptions().dtype(torch::kInt32).device(A.device()));
    //dim3 threads(1,1024);
    dim3 threads(1024);
    const dim3 blocks((C.size(0) + threads.x - 1) / threads.x, (C.size(1) + threads.y - 1) / threads.y);
    AT_DISPATCH_INTEGRAL_TYPES(B.scalar_type(), "binary_seeded_mm_cuda", ([&] {
            cuda_binary_seeded_bmv_kernel<<<blocks,threads>>>(
                    C.packed_accessor32<int32_t,2,torch::RestrictPtrTraits>(),
                    A.packed_accessor32<torch::Half,2,torch::RestrictPtrTraits>(),
                    B.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
                    bitsize,
                    seed
                    );

        }));
    return C;
}


torch::Tensor cuda_sample_bits(torch::Tensor p, int n, torch::Dtype dtype, unsigned long seed) {
    const int bitsize = 8 * elementSize(dtype);
    const int ret_size2 = ceil_div(p.size(1), bitsize);
    auto ret = torch::zeros({n, p.size(0), ret_size2}, torch::TensorOptions().dtype(dtype).device(p.device()));
    const int threads2 = next_pow2to1024(ret_size2);
    int threads0 = next_pow2(n);
        if (1024 / threads2 < threads0) {
            threads0 = 1024 / threads2;
        }
    int threads1 = 1024 / threads0 / threads2;
    if (threads1 > 64) {
        threads1 = 64;
    }
    const dim3 threads(threads0, threads2, threads1);
    const dim3 blocks(ceil_div(n, threads.x), ceil_div(ret_size2, threads.y), ceil_div(ret.size(1), threads.z));
    printf("%d %d %d\n",threads.x, threads.y, threads.z);
    printf("%d %d %d\n",blocks.x, blocks.y, blocks.z);

    AT_DISPATCH_INTEGRAL_TYPES(dtype, "sample_bits_cuda", ([&] {
                cuda_sample_bits_kernel<<<blocks,threads>>>(
                            ret.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                            p.packed_accessor32<torch::Half,2,torch::RestrictPtrTraits>(),
                            bitsize,
                            seed);

            }));
    return ret;
}

torch::Tensor cuda_binary_weighted_sum(torch::Tensor input, torch::Tensor weights, int z_bits) {
    const int bitsize = 8 * elementSize(input.scalar_type());
    const int threadsz = next_pow2to1024(z_bits);
    int threadsy = next_pow2(input.size(1));
    if (1024 / threadsz < threadsy) {
        threadsy = 1024 / threadsz;
    }
    const dim3 threads(threadsy, threadsz);
    const dim3 blocks(ceil_div(input.size(1), threadsy), ceil_div(z_bits, threadsz));
    auto ret = torch::zeros({input.size(1), z_bits}, torch::TensorOptions().dtype(torch::kFloat16).device(input.device()));
    AT_DISPATCH_INTEGRAL_TYPES(input.scalar_type(), "binary_weighted_sum_cuda", ([&] {
        cuda_binary_weighted_sum_kernel<<<blocks,threads>>>(
            ret.packed_accessor32<torch::Half,2,torch::RestrictPtrTraits>(),
            input.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
            weights.packed_accessor32<torch::Half,1,torch::RestrictPtrTraits>(),
            bitsize);
                           }));
    return ret;
}