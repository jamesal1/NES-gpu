#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_fp16.h>



//can't use __clz
int next_pow2_clip(int v, int cap) {
    if (v > cap / 2)
        return cap;
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    v++;
    return v;
}

int ceil_div (int a, int b) {
    return (a + b - 1) / b;

}


__global__ void cuda_pack8_kernel(torch::PackedTensorAccessor32<int8_t,2,torch::RestrictPtrTraits> ret,
                                 const torch::PackedTensorAccessor32<bool,2,torch::RestrictPtrTraits> input) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int y_input = y * 8;
    if (x < input.size(0)) {

        int end = input.size(1) - y_input;
        int8_t tmp = 0;
        if (end>7) {
            end = 8;
        }
        int c = 1;
        for (int i = 0; i < end; i++) {
            tmp += c * input[x][y_input+i];
            c*=2;
        }
        ret[x][y] = tmp;
    }
}


//slower than non-templated version for some reason
template <typename scalar_t>
__global__ void cuda_pack_kernel(torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> ret,
                                 const torch::PackedTensorAccessor32<bool,2,torch::RestrictPtrTraits> input,
                                 const int elementSize) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int y_input = y * elementSize;
    if (x < ret.size(0) && y < ret.size(1)) {
        scalar_t c = 1;
        int end = input.size(1) - y_input;
        if (end > elementSize) {
            end = elementSize;
        }
        scalar_t tmp = 0;
        for (int i = 0; i < end; i++) {
            tmp |= c * input[x][y_input+i];
            c *= 2;
        }
        ret[x][y] = tmp;
    }
}


template <typename scalar_t>
__global__ void cuda_unpack_kernel(torch::PackedTensorAccessor32<bool,2,torch::RestrictPtrTraits> ret,
                               torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> input, int elementSize) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y;
    const int y_output = y * elementSize;
    if (x < input.size(0)) {
            unsigned long long int c = 1;
            for (int i = 0; i < elementSize; i++) {
                ret[x][y_output+i] = (c & input[x][y]) > 0;
                c *= 2;
            }
        }
}



//there's a bug causing an occasional element to not be set
template <typename scalar_t>
__global__ void cuda_sample_bits_kernel(torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> ret,
                                        const torch::PackedTensorAccessor32<torch::Half,2,torch::RestrictPtrTraits> input,
                                        const int elementSize,
                                        const unsigned long seed) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    const int y = blockIdx.z * blockDim.z + threadIdx.z;
    const int seq = x * ret.size(1) * ret.size(2) + y * ret.size(2)  + z;
    hiprandState state;
    hiprand_init(seed + seq, 0, 0, &state);
    const int z_input = z * elementSize;
    if (x < ret.size(0) && y < ret.size(1) && z < ret.size(2)) {
        scalar_t c = 1;
        int end = input.size(1) - z_input;
        if (end > elementSize) {
            end = elementSize;
        }
        scalar_t tmp = 0;
        for (int i = 0; i < end; i++) {
            tmp |= c * ( __half2float(input[y][z_input+i]) > hiprand_uniform(&state));
            //tmp |= c * ( (input[y][z_input+i]) > hiprand_normal(&state));
            //tmp |= c * ( (input[y][z_input+i]) > .5);
            c *= 2;
        }
        ret[x][y][z] = tmp;
    }
}

//there's a bug affecting the last element somewhere
template <typename scalar_t>
__global__ void cuda_binary_weighted_sum_kernel(torch::PackedTensorAccessor32<torch::Half,2,torch::RestrictPtrTraits> ret,
                                    const torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> input,
                                    const torch::PackedTensorAccessor32<torch::Half,1,torch::RestrictPtrTraits> weights,
                                    const int elementSize) {
    //const int y = blockIdx.y * blockDim.y + threadIdx.y;
    //const int z = blockIdx.z * blockDim.z + threadIdx.z;
    const int y = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (y < ret.size(0) && z < ret.size(1)) {
        const int z_input = z / elementSize;
        const int z_bit = z - z_input * elementSize;
        scalar_t c = 1;
        c <<= z_bit;
        torch::Half tmp(0);
        for (int i = 0; i < input.size(0); i++) {
            tmp += weights[i] * ((input[i][y][z_input] & c) >> z_bit);
        }
        ret[y][z] = tmp;
    }
}


torch::Tensor cuda_pack8(torch::Tensor input) {
    const int ret_size1 = (input.size(1) + 7) / 8;
    const int threadsy = next_pow2_clip(ret_size1, 1024);
    const dim3 threads(1024 / threadsy, threadsy);
    const dim3 blocks(ceil_div(input.size(0), threads.x), ceil_div(ret_size1, threadsy));
    auto ret = torch::zeros({input.size(0), ret_size1}, torch::TensorOptions().dtype(torch::kInt8).device(input.device()));
    cuda_pack8_kernel<<<blocks,threads>>>(
        ret.packed_accessor32<int8_t,2,torch::RestrictPtrTraits>(),
        input.packed_accessor32<bool,2,torch::RestrictPtrTraits>());
    return ret;
}




torch::Tensor cuda_pack(torch::Tensor input, torch::Dtype dtype) {
    const int bitsize = 8 * elementSize(dtype);
    const int ret_size1 = (input.size(1) + bitsize - 1) / bitsize;
    const int threadsy = next_pow2_clip(ret_size1, 1024);
    const dim3 threads(1024 / threadsy, threadsy);
    const dim3 blocks(ceil_div(input.size(0), threads.x), ceil_div(ret_size1, threadsy));
    auto ret = torch::zeros({input.size(0), ret_size1}, torch::TensorOptions().dtype(dtype).device(input.device()));
    AT_DISPATCH_INTEGRAL_TYPES(ret.scalar_type(), "pack_cuda", ([&] {
        cuda_pack_kernel<<<blocks,threads>>>(
            ret.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
            input.packed_accessor32<bool,2,torch::RestrictPtrTraits>(),
            bitsize);
    }));
    return ret;
}




torch::Tensor cuda_unpack(torch::Tensor input) {
    int bitsize = 8 * elementSize(input.scalar_type());
    const int threads = 1024;
    const dim3 blocks((input.size(0) + threads - 1) / threads, input.size(1));
    auto ret = torch::zeros({input.size(0), input.size(1) * bitsize}, torch::TensorOptions().dtype(torch::kBool).device(input.device()));
    AT_DISPATCH_INTEGRAL_TYPES(input.scalar_type(), "unpack_cuda", ([&] {
        cuda_unpack_kernel<<<blocks,threads>>>(
                    ret.packed_accessor32<bool,2,torch::RestrictPtrTraits>(),
                    input.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
                    bitsize);

    }));
    return ret;
}



torch::Tensor cuda_sample_bits(torch::Tensor p, int n, torch::Dtype dtype, unsigned long seed) {
    const int bitsize = 8 * elementSize(dtype);
    const int ret_size2 = ceil_div(p.size(1), bitsize);
    auto ret = torch::zeros({n, p.size(0), ret_size2}, torch::TensorOptions().dtype(dtype).device(p.device()));
    const int threads2 = next_pow2_clip(ret_size2, 1024);
    const int threads0 = next_pow2_clip(n, 1024 / threads2);
    int threads1 = 1024 / threads0 / threads2;
    if (threads1 > 64) {
        threads1 = 64;
    }
    const dim3 threads(threads0, threads2, threads1);
    const dim3 blocks(ceil_div(n, threads.x), ceil_div(ret_size2, threads.y), ceil_div(ret.size(1), threads.z));
    //printf("%d %d %d\n",threads.x, threads.y, threads.z);
    //printf("%d %d %d\n",blocks.x, blocks.y, blocks.z);

    AT_DISPATCH_INTEGRAL_TYPES(dtype, "sample_bits_cuda", ([&] {
                cuda_sample_bits_kernel<<<blocks,threads>>>(
                            ret.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
                            p.packed_accessor32<torch::Half,2,torch::RestrictPtrTraits>(),
                            bitsize,
                            seed);

            }));
    return ret;
}

torch::Tensor cuda_binary_weighted_sum(torch::Tensor input, torch::Tensor weights, int z_bits) {
    const int bitsize = 8 * elementSize(input.scalar_type());
    const int threadsz = next_pow2_clip(z_bits, 1024);
    int threadsy = next_pow2_clip(input.size(1), 1024 / threadsz);
    const dim3 threads(threadsy, threadsz);
    const dim3 blocks(ceil_div(input.size(1), threadsy), ceil_div(z_bits, threadsz));
    auto ret = torch::zeros({input.size(1), z_bits}, torch::TensorOptions().dtype(torch::kFloat16).device(input.device()));
    AT_DISPATCH_INTEGRAL_TYPES(input.scalar_type(), "binary_weighted_sum_cuda", ([&] {
        cuda_binary_weighted_sum_kernel<<<blocks,threads>>>(
            ret.packed_accessor32<torch::Half,2,torch::RestrictPtrTraits>(),
            input.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
            weights.packed_accessor32<torch::Half,1,torch::RestrictPtrTraits>(),
            bitsize);
                           }));
    return ret;
}

