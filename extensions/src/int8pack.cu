
#include <hip/hip_runtime.h>
#define ELEMENT_SIZE 64
#define BLOCK_SIZE 16
extern "C"
__global__ void int8pack_kernel(long *ret,  const unsigned char *input, const int ret0, const int ret1, const int input1) {
    const int y = blockIdx.x * blockDim.x + threadIdx.x;
    const int x = blockIdx.y * blockDim.y + threadIdx.y;
    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int cache1 = ELEMENT_SIZE / 8 * blockDim.x;
    const int square_size = BLOCK_SIZE * BLOCK_SIZE;
    const int offset_y = blockIdx.x * blockDim.x * ELEMENT_SIZE / 8;
    const int offset = blockIdx.y * blockDim.y * input1 + offset_y;
    __shared__ unsigned char cache[BLOCK_SIZE * BLOCK_SIZE * ELEMENT_SIZE / 8];
    for (int i = 0; i < ELEMENT_SIZE / 8; i++){
        const int idx = i * square_size + tid;
        const int x_sub = idx / cache1;
        const int y_sub = idx - x_sub * cache1;
        cache[x_sub * cache1 + y_sub] = y_sub + offset_y < input1 ? input[x_sub * input1 + y_sub + offset] : 0;

    }
    __syncthreads();
    const int y_input = threadIdx.x * ELEMENT_SIZE / 8;
    if (x < ret0 && y < ret1) {
        long tmp = 0;
        #pragma unroll
        for (int i = 0; i < ELEMENT_SIZE / 8; i++) {
            tmp |= ((long) cache[threadIdx.y * cache1 + y_input + i] ) << (8 * i);
        }
        ret[x * ret1 + y] = tmp;
    }
}